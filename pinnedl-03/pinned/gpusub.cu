#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <iostream>
#include <math.h>

// GPU Kernel
__global__ void subtractVectorGPUKernel( float* ad, float* bd, float* cd, int size ){

	// Retrieve our coordinates in the block
	int tx = blockIdx.x * blockDim.x + threadIdx.x;


	// Perform
	if(tx<size){
	cd[tx]=ad[tx] - bd[tx];
	}

	
}

bool subtractVectorGPU( float* a, float* b, float* c, int size ){

	// Error return value
	hipError_t status;

	// Number of bytes in the matrix.
	int bytes = size * sizeof(float);

	// Pointers to the device arrays
	float *ad, *bd, *cd;

	// Allocate memory on the device to store each matrix
	hipHostGetDevicePointer( (void**)&ad, a, 0 );
	hipHostGetDevicePointer( (void**)&bd, b, 0 );
	hipHostGetDevicePointer( (void**)&cd, c, 0 );
	// Specify the size of the grid and the size of the block
	float dimBlock= 1024; 
	float x = (size/dimBlock);
	int dimGrid = (int)ceil(x);

	// Launch the kernel on a size-by-size block of threads
	subtractVectorGPUKernel<<<dimGrid, dimBlock>>>(ad, bd, cd, size);
	// Wait for completion
	hipDeviceSynchronize();

	// Check for errors
	status = hipGetLastError();
	if (status != hipSuccess) {
	std::cout << "Kernel failed: " <<
	hipGetErrorString(status) << std::endl;
	return false;
	}

	// Success
	return true;
}