#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <iostream>
#include <math.h>

// GPU Kernel
__global__ void scaleVectorGPUKernel( float* ad, float* cd, float scaleFactor, int size ){

	// Retrieve our coordinates in the block
	int tx = blockIdx.x * blockDim.x + threadIdx.x;


	// Perform
	if(tx<size){
	cd[tx]=ad[tx] * scaleFactor;
	}

}

bool scaleVectorGPU( float* a, float* c, float scaleFactor, int size ){

	// Error return value
	hipError_t status;

	// Number of bytes in the matrix.
	int bytes = size * sizeof(float);

	// Pointers to the device arrays
	float *ad,*cd;

	// Allocate memory on the device to store each matrix
	hipMalloc((void**) &ad, bytes);
	//hipMalloc((void**) &bd, bytes);
	hipMalloc((void**) &cd, bytes);

	// Copy the host input data to the device
	hipMemcpy(ad, a, bytes, hipMemcpyHostToDevice);
	//hipMemcpy(bd, b, bytes, hipMemcpyHostToDevice);

	
	
	// Specify the size of the grid and the size of the block
	float dimBlock= 1024; // Matrix is contained in a block
	float x = (size/dimBlock);
	int dimGrid = (int)ceil(x);// Only using a single grid element today

	// Launch the kernel on a size-by-size block of threads
	scaleVectorGPUKernel<<<dimGrid, dimBlock>>>(ad, cd, scaleFactor, size);
	// Wait for completion
	hipDeviceSynchronize();
	// Retrieve the result matrix
	hipMemcpy(c, cd, bytes, hipMemcpyDeviceToHost);

		// Check for errors
	status = hipGetLastError();
	if (status != hipSuccess) {
	std::cout << "Kernel failed: " <<
	hipGetErrorString(status) << std::endl;
	hipFree(ad);
	//hipFree(bd);
	hipFree(cd);
	return false;
	}

	


	// Free device memory
	hipFree(ad);
	//hipFree(bd);
	hipFree(cd);


	// Success
	return true;
}